#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hip/hip_runtime.h>
#include <hipsparse.h>         // hipsparseSpMV
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE
#include <fstream>
#include <iostream>
#include <random>
#include <chrono>
#include <string>
#include "GEAM.h"
#define BLOCK_SIZE 512
// Kernel for reducing a vector
__global__ void reduce_sum_float(float *d_out, float *d_in, unsigned int size) {
    // Calculate thread ID
    unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int tid = threadIdx.x;
    __shared__ float sdata[BLOCK_SIZE];
    sdata[tid] = (idx < size) ? d_in[idx] : 0;
    // Boundary check
    if (idx >= size) {
        return;
    }
    // printf("idx: %d d_in: %f\n", idx, d_in[idx]);
    // Shared memory for this block

    // printf("size: %d tid: %d idx: %d sdata[8]: %f\n", size, tid, idx, sdata[8]);

    __syncthreads(); // Make sure the entire block is loaded!

    // Do reduction in shared mem
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            // printf("s: %d tid: %d sdata[tid]: %f sdata[tid + s]: %f\n", s, tid, sdata[tid], sdata[tid + s]);
            // printf("sdata[8]: %f\n", sdata[8]);
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads(); // Make sure all additions at one stage are done!
    }

    // Only thread 0 writes result for this block back to global mem
    if (tid == 0) {
        // printf("sdata[0]: %f\n", sdata[0]);
        d_out[blockIdx.x] = sdata[0];
    }
}

__global__ void reduce_sum_float_res(float *d_out, float *d_in, unsigned int size, float* res, float* mc, float total_weights) {
    // Calculate thread ID
    unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int tid = threadIdx.x;
    __shared__ float sdata[BLOCK_SIZE];
    sdata[tid] = (idx < size) ? d_in[idx] : 0;
    // Boundary check
    if (idx >= size) {
        return;
    }
    // printf("idx: %d d_in: %f\n", idx, d_in[idx]);
    // Shared memory for this block

    // printf("size: %d tid: %d idx: %d sdata[8]: %f\n", size, tid, idx, sdata[8]);

    __syncthreads(); // Make sure the entire block is loaded!

    // Do reduction in shared mem
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            // printf("s: %d tid: %d sdata[tid]: %f sdata[tid + s]: %f\n", s, tid, sdata[tid], sdata[tid + s]);
            // printf("sdata[8]: %f\n", sdata[8]);
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads(); // Make sure all additions at one stage are done!
    }

    // Only thread 0 writes result for this block back to global mem
    if (tid == 0) {
        // printf("sdata[0]: %f\n", sdata[0]);
        d_out[blockIdx.x] = sdata[0];
        *res = -0.5 * sdata[0];
        *mc = total_weights + (-0.5) * (*res);
    }
}


void reduce_sum_float_h(float *d_in, float *out, unsigned int size) {
    // Set up execution parameters
    dim3 block(BLOCK_SIZE);
    dim3 grid((size + block.x - 1) / block.x);
    float *d_out, *d_tmp_in, *d_tmp_out;
    
    // Allocate device memory for the intermediate results
    hipMalloc((void **)&d_tmp_in, size * sizeof(float));
    hipMalloc((void **)&d_tmp_out, grid.x * sizeof(float));
    
    // Copy the input data to the temporary input buffer
    hipMemcpy(d_tmp_in, d_in, size * sizeof(float), hipMemcpyDeviceToDevice);
    
    while (size > 1) {
        // Call the reduction kernel
        reduce_sum_float<<<grid, block, block.x * sizeof(float)>>>(d_tmp_out, d_tmp_in, size);
        
        // Swap the input and output buffers for the next iteration
        d_out = d_tmp_in;
        d_tmp_in = d_tmp_out;
        d_tmp_out = d_out;
        
        // Calculate the size of the data for the next kernel launch
        size = (size + block.x - 1) / block.x;
        
        // Calculate the grid size for the next kernel launch
        grid.x = (size + block.x - 1) / block.x;
    }
    
    // Now that the data is small enough, perform the final reduction on the CPU
    float h_out;
    hipMemcpy(&h_out, d_tmp_in, sizeof(float), hipMemcpyDeviceToHost);
    
    *out = h_out;
    
    // Free the device memory
    hipFree(d_tmp_in);
    hipFree(d_tmp_out);
}

void reduce_sum_float_h(float *out, unsigned int size, float* d_tmp_in, float* d_tmp_out) {
    // Set up execution parameters
    dim3 block(BLOCK_SIZE);
    dim3 grid((size + block.x - 1) / block.x);
    float *d_out;
    
    // Allocate device memory for the intermediate results
    // hipMalloc((void **)&d_tmp_in, size * sizeof(float));
    // hipMalloc((void **)&d_tmp_out, grid.x * sizeof(float));
    
    // Copy the input data to the temporary input buffer
    // hipMemcpy(d_tmp_in, d_in, size * sizeof(float), hipMemcpyDeviceToDevice);
    
    while (size > 1) {
        // Call the reduction kernel
        reduce_sum_float<<<grid, block, block.x * sizeof(float)>>>(d_tmp_out, d_tmp_in, size);
        
        // Swap the input and output buffers for the next iteration
        d_out = d_tmp_in;
        d_tmp_in = d_tmp_out;
        d_tmp_out = d_out;
        
        // Calculate the size of the data for the next kernel launch
        size = (size + block.x - 1) / block.x;
        
        // Calculate the grid size for the next kernel launch
        grid.x = (size + block.x - 1) / block.x;
    }
    
    // Now that the data is small enough, perform the final reduction on the CPU
    float h_out;
    hipMemcpy(&h_out, d_tmp_in, sizeof(float), hipMemcpyDeviceToHost);
    
    *out = h_out;
    
    // Free the device memory
    // hipFree(d_tmp_in);
    // hipFree(d_tmp_out);
}

void reduce_sum_float_h(unsigned int size, float* d_tmp_in, float* d_tmp_out, float* energy, float* mc, float total_weights) {
    // Set up execution parameters
    dim3 block(BLOCK_SIZE);
    dim3 grid((size + block.x - 1) / block.x);
    float *d_out;
    
    // Allocate device memory for the intermediate results
    // hipMalloc((void **)&d_tmp_in, size * sizeof(float));
    // hipMalloc((void **)&d_tmp_out, grid.x * sizeof(float));
    
    // Copy the input data to the temporary input buffer
    // hipMemcpy(d_tmp_in, d_in, size * sizeof(float), hipMemcpyDeviceToDevice);
    
    while (size > 1) {
        // Call the reduction kernel
        reduce_sum_float_res<<<grid, block, block.x * sizeof(float)>>>(d_tmp_out, d_tmp_in, size, energy, mc, total_weights);
        
        // Swap the input and output buffers for the next iteration
        d_out = d_tmp_in;
        d_tmp_in = d_tmp_out;
        d_tmp_out = d_out;
        
        // Calculate the size of the data for the next kernel launch
        size = (size + block.x - 1) / block.x;
        
        // Calculate the grid size for the next kernel launch
        grid.x = (size + block.x - 1) / block.x;
    }
    
    // Free the device memory
    // hipFree(d_tmp_in);
    // hipFree(d_tmp_out);
}

#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

__global__ void calculate_sol(float* d_x, float* d_sol, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        d_sol[i] = (d_x[i] > 0) ? 1 : -1;
    }
}
// CUDA kernel to update vectors x and y
__global__ void update_vectors(float* d_x, float* d_y, float* d_jx, float p, float xi, int N, float dt) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        // printf("1 i: %d dx: %f dy: %f\n", i, d_x[i], d_y[i]);
        float x = d_x[i];
        float y = d_y[i];
        y += ((-1 + p) * x + xi * d_jx[i]) * dt;
        x += y * dt;
        // printf("2 i: %d dx: %f dy: %f\n", i, x, y);
        y = (abs(x) > 1) ? 0 : y;
        x = min(max(x, -1.0f), 1.0f);
        d_x[i] = x;
        d_y[i] = y;
        // printf("3 i: %d dx: %f dy: %f\n", i, d_x[i], d_y[i]);
    }
}

// CUDA kernel to compute the energy
// Kernel to compute energy and mc
__global__ void compute_energy_s1(float* d_sol_J, float* d_sol, int N) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i < N){
    // printf("i: %d d_sol_J: %f\n", i, d_sol_J[i]);
    d_sol_J[i] = d_sol_J[i] * d_sol[i]; 
    // printf("2 i: %d d_sol_J: %f\n", i, d_sol_J[i]);
  }
}

// CUDA kernel to square a vector and save the result in another vector
__global__ void square_and_save(const float* d_v, float* d_squared, int nnz) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < nnz) {
        d_squared[i] = d_v[i] * d_v[i];
    }
}

__global__ void reduce_sum(float* d_v, float* d_sum, int nnz) {
    extern __shared__ float sdata[];
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + tid;
    sdata[tid] = (i < nnz) ? d_v[i] : 0;
    __syncthreads();
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
    if (tid == 0) {
        d_sum[blockIdx.x] = sdata[0];
    }
}

// Main function
int main_pre(std::string s) {
    // Parameters
    int N;
    const int num_iters = 1000;
    const float dt = 0.75;
    // Load the sparse matrix from a COO file
    int nnz;
    int unit_length;
    int* cooRowInd;
    int* cooColInd;
    float* cooVal;


    std::ifstream file(s);
    if (!file.is_open()) {
        return 1;
    }
    std::cout << s;

    file >> unit_length >> nnz;

    cooRowInd = new int[nnz];
    cooColInd = new int[nnz];
    cooVal = new float[nnz];

    // Read the data
    for (int i = 0; i < nnz; i++) {
        file >> cooRowInd[i] >> cooColInd[i] >> cooVal[i];
        cooRowInd[i]--; // Adjust to 0-based indexing
        cooColInd[i]--;
        cooVal[i] = -cooVal[i]; // J = -J
    }

    //print "cooRowInd, cooColInd, cooVal"
    // std::cout << "print: " << std::endl;
    // for(int i = 0; i < nnz; i++){
    //     std::cout << cooRowInd[i] << " ";
    // }
    // std::cout << std::endl;
    // for(int i = 0; i < nnz; i++){
    //     std::cout << cooColInd[i] << " ";
    // }
    // std::cout << std::endl;
    // for(int i = 0; i < nnz; i++){
    //     std::cout << cooVal[i] << " ";
    // }
    // std::cout << std::endl;
    file.close();

    N = unit_length;

    int* cooRowInd_new;
    int* cooColInd_new;
    float* cooVal_new;
    int new_size;
    add_with_transpose(cooRowInd, cooColInd, cooVal, nnz, cooRowInd_new, cooColInd_new, cooVal_new, new_size);
    //print "cooColInd_new, cooVal_new, cooVal_new"
    // std::cout << "print: " << std::endl;
    // for(int i = 0; i < new_size; i++){
    //     std::cout << cooRowInd_new[i] << " ";
    // }
    // std::cout << std::endl;
    // for(int i = 0; i < new_size; i++){
    //     std::cout << cooColInd_new[i] << " ";
    // }
    // std::cout << std::endl;
    // for(int i = 0; i < new_size; i++){
    //     std::cout << cooVal_new[i] << " ";
    // }
    // std::cout << std::endl;
    // Allocate memory for vectors and matrix
    auto start_vec = std::chrono::system_clock::now();
    float *d_x, *d_y;
    hipMalloc(&d_x, N * sizeof(float));
    hipMalloc(&d_y, N * sizeof(float));
    hipDeviceSynchronize();
    auto end_vec = std::chrono::system_clock::now();
    auto duration_vec = std::chrono::duration_cast<std::chrono::microseconds>(end_vec - start_vec);
    std::cout << ", " << double(duration_vec.count()) * std::chrono::microseconds::period::num / std::chrono::microseconds::period::den;
    // std::cout <<  "花费了"  << double(duration_vec.count()) * std::chrono::microseconds::period::num / std::chrono::microseconds::period::den << "秒" << std::endl;
    // Initialize x and y
    // For the sake of this example, we'll just set all elements to 1
    std::default_random_engine generator(std::chrono::system_clock::now().time_since_epoch().count());
    std::uniform_real_distribution<float> distribution(-0.1, 0.1);
    float* init_x = new float[N];
    float* init_y = new float[N];
    for (int i = 0; i < N; ++i) {
        init_x[i] = distribution(generator);
        init_y[i] = distribution(generator);
    }
    // print first 10 elements of the vector
    // std::cout << "print init_x: " << std::endl;
    // for(int i = 0; i < 10; i++){
    //     std::cout << init_x[i] << " ";
    // }
    // std::cout << std::endl;
    // std::cout << "print init_y: " << std::endl;
    // for(int i = 0; i < 10; i++){
    //     std::cout << init_y[i] << " ";
    // }
    // std::cout << std::endl;
    /*For debug fix x, y
    */
    // init_x[0] = -0.099998;
    // init_x[1] = 0.051121;
    // init_x[2] = 0.006553;
    // init_x[3] = -0.090591;
    // init_x[4] = 0.035859;
    // init_y[0] = 0.092351;
    // init_y[1] = 0.0800000;
    // init_y[2] = 0.070000;
    // init_y[3] = -0.060000;
    // init_y[4] = -0.032974;
    auto start_1 = std::chrono::system_clock::now();
    hipMemcpy(d_x, init_x, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, init_y, N * sizeof(float), hipMemcpyHostToDevice);

    int* d_cooRowInd_new, *d_cooColInd_new;
    float* d_cooVal_new;
    hipMalloc(&d_cooRowInd_new, new_size * sizeof(int));
    hipMalloc(&d_cooColInd_new, new_size * sizeof(int));
    hipMalloc(&d_cooVal_new, new_size * sizeof(float));
    hipMemcpy(d_cooRowInd_new, cooRowInd_new, new_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_cooColInd_new, cooColInd_new, new_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_cooVal_new, cooVal_new, new_size * sizeof(float), hipMemcpyHostToDevice);
    // Create cuSPARSE handle
    hipsparseHandle_t handle;
    CHECK_CUSPARSE(hipsparseCreate(&handle));
    // Create sparse matrix A in COO format
    // Convert COO to CSR
    
    hipsparseSpMatDescr_t matA;

    // int* d_csrRowPtr;
    // hipMalloc((void**)&d_csrRowPtr, sizeof(int)*(N+1));


    // Create the matrix descriptor
    CHECK_CUSPARSE(hipsparseCreateCoo(&matA, N, N, new_size, d_cooRowInd_new, d_cooColInd_new, d_cooVal_new, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));
    hipDeviceSynchronize();
    auto end_1 = std::chrono::system_clock::now();
    auto duration_pre = std::chrono::duration_cast<std::chrono::microseconds>(end_1 - start_1);
    std::cout << ", " << double(duration_pre.count()) * std::chrono::microseconds::period::num / std::chrono::microseconds::period::den;
    // std::cout <<  "花费了"  << double(duration_pre.count()) * std::chrono::microseconds::period::num / std::chrono::microseconds::period::den << "秒" << std::endl;
    // // Allocate host memory for CSR row pointers
    // int* csrRowPtr = new int[N+1];

    // // Copy CSR row pointers from device to host
    // hipMemcpy(csrRowPtr, d_csrRowPtr, sizeof(int)*(N+1), hipMemcpyDeviceToHost);

    // CHECK_CUSPARSE(hipsparseCreateCoo(&matA0T, N, N, new_size, d_cooRowInd_new, d_cooColInd_new, d_cooVal_new, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));
    // // Create dense vector X and Y
    hipsparseDnVecDescr_t vecX, vecY, vecSol, vecSolJ, vecJX;
    CHECK_CUSPARSE(hipsparseCreateDnVec(&vecX, N, d_x, HIP_R_32F));
    CHECK_CUSPARSE(hipsparseCreateDnVec(&vecY, N, d_y, HIP_R_32F));



    float* d_squared;
    hipMalloc(&d_squared, new_size * sizeof(float));

    float total_weights;
    reduce_sum_float_h(d_cooVal_new, &total_weights, new_size);
    // std::cout << "Total weights: " << total_weights << std::endl;
    total_weights = -0.25 * total_weights;
    // std::cout << "Total weights: " << total_weights << std::endl;
    // Square the elements of J and save the result in d_squared
    square_and_save<<<(new_size + (BLOCK_SIZE - 1)) / BLOCK_SIZE, BLOCK_SIZE>>>(d_cooVal_new, d_squared, new_size);
    // hipDeviceSynchronize();

    // Sum the squared elements
    float sum;
    reduce_sum_float_h(d_squared, &sum, new_size);
    // Calculate xi
    float xi = 0.5f / sqrt(sum / (N - 1));
    // std::cout << "xi: " << xi << std::endl;
    // Allocate an external buffer if needed

    // Allocate memory for energies and maxcut_values
    float *energies, *maxcut_values;
    energies = new float[num_iters];
    maxcut_values = new float[num_iters];
    float* d_sol;
    float *d_sol_J;
    float *d_jx;
    hipMalloc(&d_sol, N * sizeof(float));
    hipMalloc(&d_sol_J, N * sizeof(float));
    hipMalloc(&d_jx, N * sizeof(float));
    CHECK_CUSPARSE(hipsparseCreateDnVec(&vecSol, N, d_sol, HIP_R_32F));
    CHECK_CUSPARSE(hipsparseCreateDnVec(&vecSolJ, N, d_sol_J, HIP_R_32F));
    CHECK_CUSPARSE(hipsparseCreateDnVec(&vecJX, N, d_jx, HIP_R_32F));

    // Allocate d_Jx to store the result of J @ x_comp
    float* d_Jx;  
    hipMalloc(&d_Jx, N * sizeof(float));

    float alpha        = 1.0f;
    float beta         = 0.0f;
    void*                dBuffer    = NULL;
    size_t bufferSize = 0;
    CHECK_CUSPARSE( hipsparseSpMV_bufferSize(
                                handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                &alpha, matA, vecX, &beta, vecY, HIP_R_32F,
                                HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize) )
    CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) )
    // for(int i = 0; i < num_iters; i++){
    //     std::cout << i;
    //     if( i != num_iters - 1 )
    //         std::cout << ", ";
    // }
    // std::cout << std::endl;
    // float* hY = new float[5];
    auto start = std::chrono::system_clock::now();
    // float* d_tmp_in;
    float* d_tmp_out;
    // hipMalloc((void **)&d_tmp_in, N * sizeof(float));
    hipMalloc((void **)&d_tmp_out, ((N + (BLOCK_SIZE - 1)) / BLOCK_SIZE) * sizeof(float));

    float* energy_d;
    float* mc_d;
    hipMalloc((void **)&energy_d, sizeof(float) * num_iters);
    hipMalloc((void **)&mc_d, sizeof(float) * num_iters);
    for (int i = 0; i < num_iters; i++) {
        // std::cout << "ITERATION :" << i << std::endl;
        float p = (i == 0) ? 0.0 : (i * 1.0 / static_cast<float>(num_iters - 1));
            // allocate an external buffer if needed
        // printf("p: %f\n", p);
        // Perform SpMV: J @ x_comp
        CHECK_CUSPARSE(hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecX, &beta, vecJX, HIP_R_32F, HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer));
        // hipDeviceSynchronize();
        // CHECK_CUDA( hipMemcpy(hY, d_y, 5 * sizeof(float),
        //                    hipMemcpyDeviceToHost) )
        //print hY
        // std::cout << "print hY: " << std::endl;
        // for(int j = 0; j < 5; j++){
        //     std::cout << hY[j] << " ";
        // }

        // Update x_comp and y_comp
        update_vectors<<<(N + (BLOCK_SIZE - 1)) / BLOCK_SIZE, BLOCK_SIZE>>>(d_x, d_y, d_jx, p, xi, N, dt);
        if(i == num_iters - 1){ // remove this if, you can get energy per iteration.
            calculate_sol<<<(N + (BLOCK_SIZE - 1)) / BLOCK_SIZE, BLOCK_SIZE>>>(d_x, d_sol, N);
            // Perform SpMV: sol.T @ J
            CHECK_CUSPARSE(hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecSol, &beta, vecSolJ, HIP_R_32F, HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer));
            // hipDeviceSynchronize();
            // Compute the energy and maxcut_value
            compute_energy_s1<<<(N + (BLOCK_SIZE - 1)) / BLOCK_SIZE, BLOCK_SIZE>>>(d_sol_J, d_sol, N);
            reduce_sum_float_h(N, d_sol_J, d_tmp_out, &(energy_d[i]), &(mc_d[i]), total_weights);
        }
    }
    hipDeviceSynchronize();
    auto end = std::chrono::system_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
    // std::cout <<  "花费了" << double(duration.count()) * std::chrono::microseconds::period::num / std::chrono::microseconds::period::den << "秒" << std::endl;
    std::cout << ", " << double(duration.count()) * std::chrono::microseconds::period::num / std::chrono::microseconds::period::den;

    hipMemcpy(energies, energy_d, sizeof(float) * num_iters, hipMemcpyDeviceToHost);
    hipMemcpy(maxcut_values, mc_d, sizeof(float) * num_iters, hipMemcpyDeviceToHost);
    // for(int i = 0; i < num_iters; i++){
    //     std::cout << energies[i];
    //     if( i != num_iters - 1 )
    //         std::cout << ", ";
    // }
    // std::cout << std::endl;

    // for(int i = 0; i < num_iters; i++){
    //     std::cout << maxcut_values[i];
    //     if( i != num_iters - 1 )
    //         std::cout << ", ";
    // }
    // std::cout << std::endl;
    float* h_sol = new float[N];
    auto start_sol = std::chrono::system_clock::now();
    hipMemcpy(h_sol, d_sol, N * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    auto end_sol = std::chrono::system_clock::now();
    auto duration_sol = std::chrono::duration_cast<std::chrono::microseconds>(end_sol - start_sol);
    // std::cout <<  "花费了" << double(duration_sol.count()) * std::chrono::microseconds::period::num / std::chrono::microseconds::period::den << "秒" << std::endl;
    std::cout << ", " << double(duration_sol.count()) * std::chrono::microseconds::period::num / std::chrono::microseconds::period::den << std::endl;
    // print first the vector
    // std::cout << "print h_sol: " << std::endl;
    // for(int i = 0; i < N; i++){
    //     std::cout << h_sol[i] << " ";
    // }
    // std::cout << std::endl;
// Cleanup
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_cooRowInd_new);
    hipFree(d_cooColInd_new);
    hipFree(d_cooVal_new);
    hipFree(dBuffer);
    hipsparseDestroySpMat(matA);
    hipsparseDestroyDnVec(vecX);
    hipsparseDestroyDnVec(vecY);
    hipsparseDestroy(handle);
    // hipFree(d_tmp_in);
    hipFree(d_tmp_out);
    delete[] init_x;
    delete[] init_y;
    delete[] cooRowInd;
    delete[] cooColInd;
    delete[] cooVal;
    return 0;
}

int main(){
    auto start_init = std::chrono::system_clock::now();
    hipInit(0);
    hipDeviceSynchronize();
    auto end_init = std::chrono::system_clock::now();
    auto duration_init = std::chrono::duration_cast<std::chrono::microseconds>(end_init - start_init);
    // std::cout <<  "设备启动花费了"  << double(duration_init.count()) * std::chrono::microseconds::period::num / std::chrono::microseconds::period::den << "秒" << std::endl;
    std::cout << "name, copy vector, copy matrix, compute, copy result" << std::endl;
    for (int i = 1; i <= 67; i++) {
        std::string graph_file = "./Gset/G" + std::to_string(i);
        main_pre(graph_file);
    }
}